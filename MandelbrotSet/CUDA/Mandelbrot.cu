#include "hip/hip_runtime.h"
﻿#include "Mandelbrot.h"

class gpu_complex {
private:

	double im;
	double re;

public:
	
	__device__ gpu_complex()
	{
		re = 0;
		im = 0;
	}

	__device__ gpu_complex(double real, double imaginary)
	{
		re = real;
		im = imaginary;
	}

	__device__ double real_squared()
	{
		return re * re;
	}

	__device__ double imaginary_squared()
	{
		return im * im;
	}

	__device__ double get_modulo()
	{
		return (real_squared() + imaginary_squared());
	}

	__device__ gpu_complex operator* (const gpu_complex& a)
	{
		return gpu_complex(re * a.re - im * a.im, re * im + im * a.re);
	}

	__device__ gpu_complex operator+ (const gpu_complex & a)
	{
		return gpu_complex(re + a.re, im + a.im);
	}
};

__device__ int check_iterations(int x, int y, const bool color)
{
	double s_x = (double)x / WIDTH - 1.5;
	double s_y = (double)y / HEIGHT - 0.5;
	gpu_complex z;
	gpu_complex point(s_x, s_y);
	unsigned int number_iterations = 0;
	while ((z.get_modulo() < 2) && (number_iterations <= 80))
	{
		z = z * z + point;
		number_iterations++;
	}
	if (number_iterations < 256)
	{
		return (255 * number_iterations) / 256;
	}
	return 255;
}

__global__ void calculate_mono(int* img)
{
	int x = blockIdx.x;
	int y = blockIdx.y;
	int tid = x + y * gridDim.x;

	img[tid] = check_iterations(x, y, 0);
}

void calculateMandelbrot(int* image) {
    dim3 blocksPerGrid(WIDTH, HEIGHT);
    int* dev_out;
    std::chrono::steady_clock::time_point start;
    size_t memSize;
    
	memSize = WIDTH * HEIGHT * sizeof(int);

    hipMalloc(&dev_out, memSize);
    hipMemcpy(dev_out, image, memSize, hipMemcpyHostToDevice);
    start = std::chrono::high_resolution_clock::now();
    
	calculate_mono << <blocksPerGrid, 1 >> > (dev_out);
   
    hipMemcpy(image, dev_out, memSize, hipMemcpyDeviceToHost);

    auto finish = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed = finish - start;
    std::cout << "Elapsed time: " << elapsed.count() << " s\n";
    hipFree(dev_out);
}


int main()
{

	size_t memorySize = WIDTH * HEIGHT * sizeof(int);
	int* image = (int*)malloc(memorySize);
	if (image == NULL)
		return -1;
	memset(image, 0, memorySize);
	calculateMandelbrot(image);
	ofstream img(IMG_NAME);
	if (img.is_open())
	{
		img << "P3\n" << WIDTH << " " << HEIGHT << " 255\n";
		for (int row = 0; row < WIDTH; row++)
		{
			for (int col = 0; col < HEIGHT; col++)
			{
				img << 0 << " " << image[row * WIDTH + col] << " " << image[row * WIDTH + col] << "\n";
			}
		}
		img.close();
	}
	else
	{
		cout << "Can not open the file !\n";
		exit(1);
	}
	free(image);
	return 0;
}
